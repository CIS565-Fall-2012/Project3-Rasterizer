#include "hip/hip_runtime.h"
// CIS565 CUDA Rasterizer: A simple rasterization pipeline for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <cutil_math.h>
#include <thrust/random.h>
#include "rasterizeKernels.h"
#include "rasterizeTools.h"

glm::vec3* framebuffer;
fragment* depthbuffer;
vertex* device_vboFull;
float* device_vbo;
float* device_cbo;
int* device_ibo;
int* lock;
triangle* primitives;

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
} 

//Handy dandy little hashing function that provides seeds for random number generation
__host__ __device__ unsigned int hash(unsigned int a){
    a = (a+0x7ed55d16) + (a<<12);
    a = (a^0xc761c23c) ^ (a>>19);
    a = (a+0x165667b1) + (a<<5);
    a = (a+0xd3a2646c) ^ (a<<9);
    a = (a+0xfd7046c5) + (a<<3);
    a = (a^0xb55a4f09) ^ (a>>16);
    return a;
}

//Writes a given fragment to a fragment buffer at a given location
__host__ __device__ void writeToDepthbuffer(int x, int y, fragment frag, fragment* depthbuffer, glm::vec2 resolution){
  if(x<resolution.x && y<resolution.y){
    int index = (y*resolution.x) + x;
    depthbuffer[index] = frag;
  }
}

//Reads a fragment from a given location in a fragment buffer
__host__ __device__ fragment getFromDepthbuffer(int x, int y, fragment* depthbuffer, glm::vec2 resolution){
  if(x<resolution.x && y<resolution.y){
    int index = (y*resolution.x) + x;
    return depthbuffer[index];
  }else{
    fragment f;
    return f;
  }
}

//Writes a given pixel to a pixel buffer at a given location
__host__ __device__ void writeToFramebuffer(int x, int y, glm::vec3 value, glm::vec3* framebuffer, glm::vec2 resolution){
  if(x<resolution.x && y<resolution.y){
    int index = (y*resolution.x) + x;
    framebuffer[index] = value;
  }
}

//Reads a pixel from a pixel buffer at a given location
__host__ __device__ glm::vec3 getFromFramebuffer(int x, int y, glm::vec3* framebuffer, glm::vec2 resolution){
  if(x<resolution.x && y<resolution.y){
    int index = (y*resolution.x) + x;
    return framebuffer[index];
  }else{
    return glm::vec3(0,0,0);
  }
}

//Kernel that clears a given pixel buffer with a given color
__global__ void clearImage(glm::vec2 resolution, glm::vec3* image, glm::vec3 color){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y){
      image[index] = color;
    }
}

//Kernel that clears a given fragment buffer with a given fragment
__global__ void clearDepthBuffer(glm::vec2 resolution, fragment* buffer, fragment frag, int* lock){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
	if(x<=resolution.x && y<=resolution.y){
	  lock[index] = 0;
      fragment f = frag;
      f.position.x = x;
      f.position.y = y;
      buffer[index] = f;
    }
}

//Kernel that writes the image to the OpenGL PBO directly. 
__global__ void sendImageToPBO(uchar4* PBOpos, glm::vec2 resolution, glm::vec3* image){
  
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  
  if(x<=resolution.x && y<=resolution.y){

      glm::vec3 color;      
      color.x = image[index].x*255.0;
      color.y = image[index].y*255.0;
      color.z = image[index].z*255.0;

      if(color.x>255){
        color.x = 255;
      }

      if(color.y>255){
        color.y = 255;
      }

      if(color.z>255){
        color.z = 255;
      }
      
      // Each thread writes one pixel location in the texture (textel)
      PBOpos[index].w = 0;
      PBOpos[index].x = color.x;     
      PBOpos[index].y = color.y;
      PBOpos[index].z = color.z;
  }
}

//TODO: Implement a vertex shader
__global__ void vertexShadeKernel(float* vbo, int vbosize, vertex* vboFull, glm::mat4 mM, glm::mat4 vM, glm::mat4 pM, glm::mat4 sM){
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if(index<vbosize/3)
  {
	  int idxX = index*3;
	  int idxY = idxX + 1;
	  int idxZ = idxX + 2;

	  glm::vec4 v( vbo[idxX], vbo[idxY], vbo[idxZ], 1.0f );
	  //v = v*mM*vM*pM;
	  v = mM*v;

	  vboFull[index].lightdir = ( glm::vec4( 10, 10, 10, 1 ) - v ).swizzle(glm::X, glm::Y, glm::Z);

	  v = sM*pM*vM*v;
	  
	  vboFull[index].position = v.swizzle(glm::X, glm::Y, glm::Z);
	  vboFull[index].position /= v.w;
  }
}

//TODO: Implement primative assembly
__global__ void primitiveAssemblyKernel(vertex* vbo, int vbosize, float* cbo, int cbosize, int* ibo, int ibosize, triangle* primitives, int frame){
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  int primitivesCount = ibosize/3;
  if(index<primitivesCount){
	  int idx0 = index*3;
	  int idx1 = idx0 + 1;
	  int idx2 = idx0 + 2;

	  int idxX = ibo[idx0]*3;
	  int idxY = idxX + 1;
	  int idxZ = idxX + 2;
	  
	  primitives[ index ].v0.position = vbo[ibo[idx0]].position;
	  primitives[ index ].v0.color = glm::vec3( cbo[(idxX+frame)%cbosize], cbo[(idxY+frame)%cbosize], cbo[(idxZ+frame)%cbosize] );

	  idxX = ibo[idx1]*3;
	  idxY = idxX + 1;
	  idxZ = idxX + 2;

	  primitives[ index ].v1.position = vbo[ibo[idx1]].position;
	  primitives[ index ].v1.color = glm::vec3( cbo[(idxX+frame)%cbosize], cbo[(idxY+frame)%cbosize], cbo[(idxZ+frame)%cbosize] );

	  idxX = ibo[idx2]*3;
	  idxY = idxX + 1;
	  idxZ = idxX + 2;

	  primitives[ index ].v2.position = vbo[ibo[idx2]].position;
	  primitives[ index ].v2.color = glm::vec3( cbo[(idxX+frame)%cbosize], cbo[(idxY+frame)%cbosize], cbo[(idxZ+frame)%cbosize] );
  }
}

//TODO: Implement a rasterization method, such as scanline.
__global__ void rasterizationKernel(triangle* primitives, int primitivesCount, fragment* depthbuffer, int * lock, glm::vec2 resolution){
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if(index<primitivesCount){
	  glm::vec3 p0,p1,p2;
	  

	  p0 = primitives[index].v0.position;
	  p1 = primitives[index].v1.position;
	  p2 = primitives[index].v2.position;

	  glm::vec3 rA = p1-p0;
	  glm::vec3 rB = p2-p0;

	  glm::vec3 norm = glm::normalize( glm::cross( glm::normalize( rA ), glm::normalize( rB ) ) );

	  if( norm.z <= 0 ) return; //backface culling -should try to move this to the primitive stage if time allows

	  glm::vec3 minP;
	  glm::vec3 maxP;

	  triangle tri = primitives[index];

	  getAABBForTriangle( tri, minP, maxP );

	  int dIndex;
	  int dY;

	  for( int y = minP.y; y <= maxP.y; y++ )
	  {
		  dY = y*resolution.x;
		  for( int x = minP.x; x <= maxP.x; x++ )
		  {
			  glm::vec3 barycoord = calculateBarycentricCoordinate( tri, glm::vec2( x, y ) );
			  dIndex = dY + x;
			  if( isBarycentricCoordInBounds( barycoord ) )
			  {
				  float depth = getZAtCoordinate( barycoord, tri );
				  bool inLoop = true;
				  while( inLoop )
				  {
					  if( atomicExch( &(lock[dIndex]), 1 ) == 0 )
					  {
						  if( depth > depthbuffer[dIndex].position.z )
						  {
							  depthbuffer[dIndex].position.x = x;
							  depthbuffer[dIndex].position.y = y;
							  depthbuffer[dIndex].position.z = depth;
							  depthbuffer[dIndex].color = tri.v0.color*barycoord.x + 
														  tri.v1.color*barycoord.y + 
														  tri.v2.color*barycoord.z;
							  depthbuffer[dIndex].normal = norm;
							  depthbuffer[dIndex].lightdir = tri.v0.lightdir*barycoord.x + 
															 tri.v1.lightdir*barycoord.y + 
															 tri.v2.lightdir*barycoord.z;
						  }
						  inLoop = false;
						  atomicExch( &(lock[dIndex]), 0 );
					  }
				  }
			  }
		  }
	  }
  }
}

//TODO: Implement a fragment shader
__global__ void fragmentShadeKernel(fragment* depthbuffer, glm::vec2 resolution ){
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  if(x<=resolution.x && y<=resolution.y){
	  depthbuffer[index].color *= ( float ) max( 0.0f, -glm::dot( glm::normalize( depthbuffer[index].normal ), glm::normalize( depthbuffer[index].lightdir ) ) ) + glm::vec3( 0.1f );
  }
}

//Writes fragment colors to the framebuffer
__global__ void render(glm::vec2 resolution, fragment* depthbuffer, glm::vec3* framebuffer){

  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);

  if(x<=resolution.x && y<=resolution.y){
	  framebuffer[index] = depthbuffer[index].color; 
  }
}

// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management
void cudaRasterizeCore(uchar4* PBOpos, glm::vec2 resolution, float frame, float* vbo, int vbosize, float* cbo, int cbosize, int* ibo, int ibosize){


	glm::mat4 modelMatrix = glm::rotate( glm::mat4(1.0f), frame, glm::vec3( 0, 1, 0 ) );
	glm::mat4 projectionMatrix = glm::perspective( 45.0f, resolution.x/resolution.y, 0.1f, 100.0f );
	glm::mat4 screenMatrix = glm::mat4(1.0f);
	screenMatrix = glm::scale( screenMatrix, glm::vec3( resolution, 1.0f ) );
	screenMatrix = glm::translate( screenMatrix, glm::vec3( 0.5, 0.5, 0 ) );
	screenMatrix = glm::scale( screenMatrix, -glm::vec3( 0.5, 0.5, -1 ) );
	glm::mat4 viewMatrix = glm::lookAt( glm::vec3( 0, 2, 2 ), glm::vec3( 0, 0, 0 ), glm::vec3( 0, 1, 0 ) );

  // set up crucial magic
  int tileSize = 8;
  dim3 threadsPerBlock(tileSize, tileSize);
  dim3 fullBlocksPerGrid((int)ceil(float(resolution.x)/float(tileSize)), (int)ceil(float(resolution.y)/float(tileSize)));

  //set up framebuffer
  framebuffer = NULL;
  hipMalloc((void**)&framebuffer, (int)resolution.x*(int)resolution.y*sizeof(glm::vec3));
  
  //set up depthbuffer
  depthbuffer = NULL;
  hipMalloc((void**)&depthbuffer, (int)resolution.x*(int)resolution.y*sizeof(fragment));
  lock = NULL;
  hipMalloc((void**)&lock, (int)resolution.x*(int)resolution.y*sizeof(int));

  //kernel launches to black out accumulated/unaccumlated pixel buffers and clear our scattering states
  clearImage<<<fullBlocksPerGrid, threadsPerBlock>>>(resolution, framebuffer, glm::vec3(0,0,0));
  
  fragment frag;
  frag.color = glm::vec3(0,0,0);
  frag.normal = glm::vec3(0,0,0);
  frag.position = glm::vec3(0,0,-10000);
  clearDepthBuffer<<<fullBlocksPerGrid, threadsPerBlock>>>(resolution, depthbuffer,frag,lock);

  //------------------------------
  //memory stuff
  //------------------------------
  primitives = NULL;
  hipMalloc((void**)&primitives, (ibosize/3)*sizeof(triangle));

  device_ibo = NULL;
  hipMalloc((void**)&device_ibo, ibosize*sizeof(int));
  hipMemcpy( device_ibo, ibo, ibosize*sizeof(int), hipMemcpyHostToDevice);
  
  device_vbo = NULL;
  hipMalloc((void**)&device_vbo, vbosize*sizeof(float));
  hipMemcpy( device_vbo, vbo, vbosize*sizeof(float), hipMemcpyHostToDevice);

  device_vboFull = NULL;
  hipMalloc((void**)&device_vboFull, vbosize/3*sizeof(float));

  device_cbo = NULL;
  hipMalloc((void**)&device_cbo, cbosize*sizeof(float));
  hipMemcpy( device_cbo, cbo, cbosize*sizeof(float), hipMemcpyHostToDevice);

  tileSize = 32;
  int primitiveBlocks = ceil(((float)vbosize/3)/((float)tileSize));

  //------------------------------
  //vertex shader
  //------------------------------
  vertexShadeKernel<<<primitiveBlocks, tileSize>>>(device_vbo, vbosize, device_vboFull, modelMatrix, viewMatrix, projectionMatrix, screenMatrix);

  hipDeviceSynchronize();
  //------------------------------
  //primitive assembly
  //------------------------------
  primitiveBlocks = ceil(((float)ibosize/3)/((float)tileSize));
  primitiveAssemblyKernel<<<primitiveBlocks, tileSize>>>(device_vboFull, vbosize, device_cbo, cbosize, device_ibo, ibosize, primitives, 0*(int)frame);

  hipDeviceSynchronize();
  //------------------------------
  //rasterization
  //------------------------------
  rasterizationKernel<<<primitiveBlocks, tileSize>>>(primitives, ibosize/3, depthbuffer, lock, resolution);

  hipDeviceSynchronize();
  checkCUDAError("Rasterizer");
  //------------------------------
  //fragment shader
  //------------------------------
  fragmentShadeKernel<<<fullBlocksPerGrid, threadsPerBlock>>>(depthbuffer, resolution);
  checkCUDAError("Frag Shader");

  hipDeviceSynchronize();
  //------------------------------
  //write fragments to framebuffer
  //------------------------------
  render<<<fullBlocksPerGrid, threadsPerBlock>>>(resolution, depthbuffer, framebuffer);
  sendImageToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, resolution, framebuffer);

  hipDeviceSynchronize();

  kernelCleanup();

  checkCUDAError("Kernel failed!");
}

void kernelCleanup(){
  hipFree( primitives );
  hipFree( device_vbo );
  hipFree( device_vboFull );
  hipFree( device_cbo );
  hipFree( device_ibo );
  hipFree( framebuffer );
  hipFree( depthbuffer );
}

